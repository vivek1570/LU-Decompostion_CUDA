#include "hip/hip_runtime.h"

#include "luheader.cuh"

void linearfind(double *ans, double *matL, double *matU,double *matb,double *tempd, int size){
  int i,j;
  double d,x;
  
  //forward substitution
  for (i=0;i<size;i++){
    d=matb[i];
    for (j=0;j<i;j++){
      d=d-matL[i*size+j]*tempd[j]; 
    }
    tempd[i]=d;
  }
  
  //backward substitution
  for (i=size-1;i>=0;i--){
    x=tempd[i]; 
    for (j=i+1;j<size;j++){
      x=x-matU[i*size+j]*ans[j]; 
    }
    ans[i]=x/matU[i*size+i];
  }	
  
}

int main(int argc, char *argv[]){
  int size;
  
  if(argc<4){
    fprintf(stderr,"Please enter all args eg: matA.txt ans.txt report.txt \n where the format is Ax=b ");
    exit(EXIT_FAILURE);
    }
    
  //reading files

  std::ifstream file(argv[1]);
  if (!file.is_open()) {
    std::cerr << "Error opening file!" << std::endl;
    exit(EXIT_FAILURE);}

  file >> size;
  
  double *matA=(double*)malloc(size*size*sizeof(double));
  double *matL=(double*)malloc(size*size*sizeof(double));
  double *matU=(double*)malloc(size*size*sizeof(double));
  double *matb=(double*)malloc(size*sizeof(double));
  double *tempd=(double*)malloc(size*sizeof(double));
  double *ans=(double*)malloc(size*sizeof(double));

  //read start for matrix A & B
  clock_t ra=clock(); 

  for (int i = 0; i < size * size; i++) {
      file >> matA[i];
  }
  for (int i = 0; i < size; i++) {
      file >> matb[i];
  }

  file.close();

  //time for read A &B
  clock_t re=clock(); //read end for matrix
  double rt=(double)((re-ra)/(double)CLOCKS_PER_SEC);

  // declaration of output file
  FILE *fp1;
  fp1=fopen(argv[3],"w");
  fprintf(fp1,"Time taken to read A and B matrices from file: %1.10f s\n",rt);

  

  int i;

  
  // linear equation total time with start and stop
  clock_t start=clock();

  
  ludecompose(matL,matU,matA,size,fp1);// in gpu
  
  linearfind(ans, matL, matU,matb,tempd,size); //in cpu
  
  clock_t stop=clock();
    double cputime=(double)((stop-start)/(double)CLOCKS_PER_SEC);
    fprintf(fp1,"Total time taken in solving system of linear equations for the given size: %1.10f s\n",cputime);
  
    // printarray(ans,size,1);

  //writing to ans.txt file
  FILE *fp2;
  fp2=fopen(argv[2],"w");
  // fprintf(fp2,"size:\n");
  fprintf(fp2,"%d\n",size);

  // fprintf(fp2,"\nLower triangular matrix:\n\n");
  for(i=0;i<(size);i++)
  {
    for(int j=0;j<size;j++)
    {
      fprintf(fp2,"%f ",matL[i*size+j]);
    }
    fprintf(fp2,"\n");
  }

  // fprintf(fp2,"\nUpper triangular matrix:\n\n");
  fprintf(fp2,"\n");
  for(i=0;i<(size);i++)
  {
    for(int j=0;j<size;j++)
    {
      fprintf(fp2,"%f ",matU[i*size+j]);
    }
    fprintf(fp2,"\n");
  }

  // fprintf(fp2,"\nSolution matrix:\n\n");
  for (i=0;i<size;i++){
    fprintf(fp2,"%.10f ",ans[i]);
    fprintf(fp2,"\n");
  }	
  fclose(fp2);	
  fclose(fp1);
  
  
  return 0;

}